// basic_streams.cu -- CUDA 13.0 stream overlap demo with error handling.

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

__global__ void scale_kernel(float* data, int n, float scale) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    data[idx] = data[idx] * scale + 0.001f;
  }
}

int main() {
  constexpr int N = 1 << 20;
  constexpr size_t BYTES = N * sizeof(float);

  float *h_a = nullptr, *h_b = nullptr;
  CUDA_CHECK(hipHostMalloc(&h_a, BYTES, hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc(&h_b, BYTES, hipHostMallocDefault));
  for (int i = 0; i < N; ++i) {
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  float *d_a = nullptr, *d_b = nullptr;
  CUDA_CHECK(hipMalloc(&d_a, BYTES));
  CUDA_CHECK(hipMalloc(&d_b, BYTES));

  hipStream_t stream1 = nullptr, stream2 = nullptr;
  CUDA_CHECK(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
  CUDA_CHECK(hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, 0));

  CUDA_CHECK(hipMemcpyAsync(d_a, h_a, BYTES, hipMemcpyHostToDevice, stream1));
  CUDA_CHECK(hipMemcpyAsync(d_b, h_b, BYTES, hipMemcpyHostToDevice, stream2));

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  scale_kernel<<<grid, block, 0, stream1>>>(d_a, N, 1.1f);
  scale_kernel<<<grid, block, 0, stream2>>>(d_b, N, 0.9f);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipMemcpyAsync(h_a, d_a, BYTES, hipMemcpyDeviceToHost, stream1));
  CUDA_CHECK(hipMemcpyAsync(h_b, d_b, BYTES, hipMemcpyDeviceToHost, stream2));

  CUDA_CHECK(hipStreamSynchronize(stream1));
  CUDA_CHECK(hipStreamSynchronize(stream2));

  std::printf("stream1 result: %.3f\n", h_a[0]);
  std::printf("stream2 result: %.3f\n", h_b[0]);

  CUDA_CHECK(hipStreamDestroy(stream1));
  CUDA_CHECK(hipStreamDestroy(stream2));
  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipHostFree(h_a));
  CUDA_CHECK(hipHostFree(h_b));
  return 0;
}
