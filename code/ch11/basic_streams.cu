// basic_streams.cu -- CUDA 12.9 stream overlap demo (Blackwell-ready)

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void scale_kernel(float* data, int n, float scale) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    data[idx] = data[idx] * scale + 0.001f;
  }
}

int main() {
  constexpr int N = 1 << 20;
  constexpr size_t BYTES = N * sizeof(float);

  float *h_a, *h_b;
  hipHostMalloc(&h_a, BYTES, hipHostMallocDefault);
  hipHostMalloc(&h_b, BYTES, hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  float *d_a, *d_b;
  hipMalloc(&d_a, BYTES);
  hipMalloc(&d_b, BYTES);

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  // Pipeline: copy -> compute -> copy back on each stream
  hipMemcpyAsync(d_a, h_a, BYTES, hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_b, h_b, BYTES, hipMemcpyHostToDevice, stream2);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  scale_kernel<<<grid, block, 0, stream1>>>(d_a, N, 1.1f);
  scale_kernel<<<grid, block, 0, stream2>>>(d_b, N, 0.9f);

  hipMemcpyAsync(h_a, d_a, BYTES, hipMemcpyDeviceToHost, stream1);
  hipMemcpyAsync(h_b, d_b, BYTES, hipMemcpyDeviceToHost, stream2);

  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  printf("stream1 result: %.3f\n", h_a[0]);
  printf("stream2 result: %.3f\n", h_b[0]);

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipFree(d_a);
  hipFree(d_b);
  hipHostFree(h_a);
  hipHostFree(h_b);
  return 0;
}
