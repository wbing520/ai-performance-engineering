// stream_ordered_allocator.cu -- async allocator example with error checks.

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

constexpr int N = 1 << 20;

__global__ void compute_kernel(const float* __restrict__ in,
                               float* __restrict__ out,
                               int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float val = in[idx];
    out[idx] = val * val + 1.0f;
  }
}

int main() {
  float *h_src = nullptr, *h_dst1 = nullptr, *h_dst2 = nullptr;
  CUDA_CHECK(hipHostMalloc(&h_src, N * sizeof(float), hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc(&h_dst1, N * sizeof(float), hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc(&h_dst2, N * sizeof(float), hipHostMallocDefault));
  for (int i = 0; i < N; ++i) h_src[i] = static_cast<float>(i);

  hipStream_t stream1 = nullptr, stream2 = nullptr;
  CUDA_CHECK(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
  CUDA_CHECK(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));

  float *d_in1 = nullptr, *d_out1 = nullptr;
  float *d_in2 = nullptr, *d_out2 = nullptr;
  CUDA_CHECK(hipMallocAsync(&d_in1, N * sizeof(float), stream1));
  CUDA_CHECK(hipMallocAsync(&d_out1, N * sizeof(float), stream1));
  CUDA_CHECK(hipMallocAsync(&d_in2, N * sizeof(float), stream2));
  CUDA_CHECK(hipMallocAsync(&d_out2, N * sizeof(float), stream2));

  CUDA_CHECK(hipMemcpyAsync(d_in1, h_src, N * sizeof(float), hipMemcpyHostToDevice, stream1));
  CUDA_CHECK(hipMemcpyAsync(d_in2, h_src, N * sizeof(float), hipMemcpyHostToDevice, stream2));

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  compute_kernel<<<grid, block, 0, stream1>>>(d_in1, d_out1, N);
  compute_kernel<<<grid, block, 0, stream2>>>(d_in2, d_out2, N);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipMemcpyAsync(h_dst1, d_out1, N * sizeof(float), hipMemcpyDeviceToHost, stream1));
  CUDA_CHECK(hipMemcpyAsync(h_dst2, d_out2, N * sizeof(float), hipMemcpyDeviceToHost, stream2));

  CUDA_CHECK(hipStreamSynchronize(stream1));
  CUDA_CHECK(hipStreamSynchronize(stream2));

  std::printf("stream1 result[0]=%.1f\n", h_dst1[0]);
  std::printf("stream2 result[0]=%.1f\n", h_dst2[0]);

  CUDA_CHECK(hipFreeAsync(d_in1, stream1));
  CUDA_CHECK(hipFreeAsync(d_out1, stream1));
  CUDA_CHECK(hipFreeAsync(d_in2, stream2));
  CUDA_CHECK(hipFreeAsync(d_out2, stream2));
  CUDA_CHECK(hipStreamDestroy(stream1));
  CUDA_CHECK(hipStreamDestroy(stream2));
  CUDA_CHECK(hipHostFree(h_src));
  CUDA_CHECK(hipHostFree(h_dst1));
  CUDA_CHECK(hipHostFree(h_dst2));
  return 0;
}
