// stream_ordered_allocator.cu -- CUDA 12.9 async allocation across streams.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void compute_kernel(const float* in, float* out, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float val = in[idx];
    out[idx] = val * val + 1.0f;
  }
}

int main() {
  float *h_src, *h_dst1, *h_dst2;
  hipHostMalloc(&h_src, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_dst1, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_dst2, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < N; ++i) h_src[i] = static_cast<float>(i);

  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  float *d_in1, *d_out1, *d_in2, *d_out2;
  hipMallocAsync(&d_in1, N * sizeof(float), stream1);
  hipMallocAsync(&d_out1, N * sizeof(float), stream1);
  hipMallocAsync(&d_in2, N * sizeof(float), stream2);
  hipMallocAsync(&d_out2, N * sizeof(float), stream2);

  hipMemcpyAsync(d_in1, h_src, N * sizeof(float), hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_in2, h_src, N * sizeof(float), hipMemcpyHostToDevice, stream2);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  compute_kernel<<<grid, block, 0, stream1>>>(d_in1, d_out1, N);
  compute_kernel<<<grid, block, 0, stream2>>>(d_in2, d_out2, N);

  hipMemcpyAsync(h_dst1, d_out1, N * sizeof(float), hipMemcpyDeviceToHost, stream1);
  hipMemcpyAsync(h_dst2, d_out2, N * sizeof(float), hipMemcpyDeviceToHost, stream2);

  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  printf("stream1 result[0]=%.1f\n", h_dst1[0]);
  printf("stream2 result[0]=%.1f\n", h_dst2[0]);

  hipFreeAsync(d_in1, stream1);
  hipFreeAsync(d_out1, stream1);
  hipFreeAsync(d_in2, stream2);
  hipFreeAsync(d_out2, stream2);
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipHostFree(h_src);
  hipHostFree(h_dst1);
  hipHostFree(h_dst2);
  return 0;
}
