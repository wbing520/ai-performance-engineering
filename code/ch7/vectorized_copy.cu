// vectorized_copy.cu -- vectorized global load using float4 for Chapter 7.

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

struct alignas(16) Float4 { float x, y, z, w; };

constexpr int NUM_FLOATS = 1 << 20;
static_assert(NUM_FLOATS % 4 == 0, "NUM_FLOATS must be divisible by 4");
constexpr int NUM_VEC = NUM_FLOATS / 4;

__global__ void copyVectorized(const Float4* __restrict__ in,
                               Float4* __restrict__ out,
                               int n_vec) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n_vec) {
    out[idx] = in[idx];
  }
}

int main() {
  static_assert(sizeof(Float4) == 16, "Float4 must be 16 bytes");
  Float4* h_in = nullptr;
  Float4* h_out = nullptr;
  CUDA_CHECK(hipHostMalloc(&h_in, NUM_VEC * sizeof(Float4), hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc(&h_out, NUM_VEC * sizeof(Float4), hipHostMallocDefault));
  float* h_in_f = reinterpret_cast<float*>(h_in);
  for (int i = 0; i < NUM_FLOATS; ++i) {
    h_in_f[i] = static_cast<float>(i);
  }

  Float4* d_in = nullptr;
  Float4* d_out = nullptr;
  CUDA_CHECK(hipMalloc(&d_in, NUM_VEC * sizeof(Float4)));
  CUDA_CHECK(hipMalloc(&d_out, NUM_VEC * sizeof(Float4)));
  CUDA_CHECK(hipMemcpy(d_in, h_in, NUM_VEC * sizeof(Float4), hipMemcpyHostToDevice));

  int n_vec = NUM_VEC;
  dim3 block(256);
  dim3 grid((n_vec + block.x - 1) / block.x);
  copyVectorized<<<grid, block>>>(d_in, d_out, n_vec);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(h_out, d_out, NUM_VEC * sizeof(Float4), hipMemcpyDeviceToHost));
  const float* h_out_f = reinterpret_cast<const float*>(h_out);
  std::printf("out[0]=%.1f out[last]=%.1f\n", h_out_f[0], h_out_f[NUM_FLOATS - 1]);

  CUDA_CHECK(hipFree(d_in));
  CUDA_CHECK(hipFree(d_out));
  CUDA_CHECK(hipHostFree(h_in));
  CUDA_CHECK(hipHostFree(h_out));
  return 0;
}
