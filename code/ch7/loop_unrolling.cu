// loop_unrolling.cu -- loop unrolling example with separate input/output arrays.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void kernel_unrolled(const float* in, float* out, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float val = in[idx];
#pragma unroll 4
    for (int i = 0; i < 16; ++i) {
      val = val * 1.001f + 0.001f;
    }
    out[idx] = val;
  }
}

int main() {
  float *h_in, *h_out;
  hipHostMalloc(&h_in, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < N; ++i) h_in[i] = 1.0f;

  float *d_in, *d_out;
  hipMalloc(&d_in, N * sizeof(float));
  hipMalloc(&d_out, N * sizeof(float));
  hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  kernel_unrolled<<<grid, block>>>(d_in, d_out, N);
  hipDeviceSynchronize();

  hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("out[0]=%.3f\n", h_out[0]);

  hipFree(d_in);
  hipFree(d_out);
  hipHostFree(h_in);
  hipHostFree(h_out);
  return 0;
}
