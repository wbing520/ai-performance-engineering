#include <hip/hip_runtime.h>

__global__ void copyScalar(const float* __restrict__ in, float* __restrict__ out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        // Scalar load: 4-byte copy per thread
        out[idx] = in[idx];
    }
}

int main() {
    const int N = 1 << 20;
    
    float* h_in = nullptr;
    float* h_out = nullptr;
    hipHostMalloc(&h_in, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < N; ++i) h_in[i] = float(i);
    
    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(256), grid((N + 255) / 256);
    copyScalar<<<grid, block>>>(d_in, d_out, N);
    hipDeviceSynchronize();
    
    hipFree(d_in); 
    hipFree(d_out);
    hipHostFree(h_in);
    hipHostFree(h_out);
    
    return 0;
}
