// async_prefetch_tma.cu -- simplified tiled streaming example (no TMA).

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int TILE_SIZE = 1024;

__global__ void kernel(const float* data, float* out, int tiles) {
  extern __shared__ float smem[];
  const int tid = threadIdx.x;

  for (int t = 0; t < tiles; ++t) {
    const float* tile = data + t * TILE_SIZE;
    for (int i = tid; i < TILE_SIZE; i += blockDim.x) {
      smem[i] = tile[i];
    }
    __syncthreads();

    for (int i = tid; i < TILE_SIZE; i += blockDim.x) {
      out[t * TILE_SIZE + i] = smem[i] * 2.0f;
    }
    __syncthreads();
  }
}

int main() {
  constexpr int tiles = 64;
  constexpr int total = tiles * TILE_SIZE;

  float *h_in, *h_out;
  hipHostMalloc(&h_in, total * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_out, total * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < total; ++i) h_in[i] = static_cast<float>(i);

  float *d_in, *d_out;
  hipMalloc(&d_in, total * sizeof(float));
  hipMalloc(&d_out, total * sizeof(float));
  hipMemcpy(d_in, h_in, total * sizeof(float), hipMemcpyHostToDevice);

  kernel<<<1, 256, TILE_SIZE * sizeof(float)>>>(d_in, d_out, tiles);
  hipDeviceSynchronize();

  hipMemcpy(h_out, d_out, total * sizeof(float), hipMemcpyDeviceToHost);
  printf("out[0]=%.1f\n", h_out[0]);

  hipFree(d_in);
  hipFree(d_out);
  hipHostFree(h_in);
  hipHostFree(h_out);
  return 0;
}
