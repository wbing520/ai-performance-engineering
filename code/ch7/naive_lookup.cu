// Architecture-specific optimizations for CUDA 12.9
// Targets Blackwell B200/B300 (sm_100)
#include <hip/hip_runtime.h>

#define T 1024

__global__ void naiveLookup(const float* table, float* out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        // __ldg not used here, each access goes to
        // global memory without using read-only cache
        int t = idx % T;
        out[idx] = table[t];
    }
}

int main() {
    const int N = 1 << 20;
    
    float* h_table = nullptr;
    float* h_out = nullptr;
    hipHostMalloc(&h_table, T * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < T; ++i) h_table[i] = float(i);
    
    float *d_table, *d_out;
    hipMalloc(&d_table, T * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    
    hipMemcpy(d_table, h_table, T * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(256), grid((N + 255) / 256);
    naiveLookup<<<grid, block>>>(d_table, d_out, N);
    hipDeviceSynchronize();
    
    hipFree(d_table);
    hipFree(d_out);
    hipHostFree(h_table);
    hipHostFree(h_out);
    
    return 0;
}
