// optimized_lookup.cu -- coalesced gather using int4 loads.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void lookupOptimized(const float* table, const int* indices, float* out, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    out[idx] = table[indices[idx]];
  }
}

int main() {
  float *h_table, *h_out;
  int *h_indices;
  hipHostMalloc(&h_table, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_indices, N * sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < N; ++i) {
    h_table[i] = static_cast<float>(i);
    h_indices[i] = i;
  }

  float *d_table, *d_out;
  int *d_indices;
  hipMalloc(&d_table, N * sizeof(float));
  hipMalloc(&d_indices, N * sizeof(int));
  hipMalloc(&d_out, N * sizeof(float));

  hipMemcpy(d_table, h_table, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, N * sizeof(int), hipMemcpyHostToDevice);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  lookupOptimized<<<grid, block>>>(d_table, d_indices, d_out, N);
  hipDeviceSynchronize();

  hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("out[0]=%.1f\n", h_out[0]);

  hipFree(d_table);
  hipFree(d_indices);
  hipFree(d_out);
  hipHostFree(h_table);
  hipHostFree(h_indices);
  hipHostFree(h_out);
  return 0;
}
