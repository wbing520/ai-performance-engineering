#include <hip/hip_runtime.h>

#define T 1024

__global__ void lookup(const float* __restrict__ table,
                      float* out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        int t = idx % T;
        // Compiler will turn this into an LDG load from the read-only cache for faster loads
        out[idx] = table[t];
    }
}

int main() {
    const int N = 1 << 20;
    
    float* h_table = nullptr;
    float* h_out = nullptr;
    hipHostMalloc(&h_table, T * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < T; ++i) h_table[i] = float(i);
    
    float *d_table, *d_out;
    hipMalloc(&d_table, T * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    
    hipMemcpy(d_table, h_table, T * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 block(256), grid((N + 255) / 256);
    lookup<<<grid, block>>>(d_table, d_out, N);
    hipDeviceSynchronize();
    
    hipFree(d_table);
    hipFree(d_out);
    hipHostFree(h_table);
    hipHostFree(h_out);
    
    return 0;
}
