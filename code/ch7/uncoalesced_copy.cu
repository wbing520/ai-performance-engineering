// Architecture-specific optimizations for CUDA 12.9
// Supports Hopper H100/H200 (sm_90) and Blackwell B200/B300 (sm_100)
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void uncoalescedCopy(const float* __restrict__ in, float* __restrict__ out, int N, int stride) {
    // n = 1048576, stride = 2
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        // Loads from in[] with a stride, causing
        // multiple memory segments to be fetched
        out[idx] = in[idx * stride];
    }
}

int main() {
    const int N = 1 << 20;
    const int stride = 2;
    
    float* h_in = nullptr;
    float* h_out = nullptr;
    hipHostMalloc(&h_in, N * stride * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < N * stride; ++i) {
        h_in[i] = static_cast<float>(i);
    }
    
    float *d_in, *d_out;
    hipMalloc(&d_in, N * stride * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    
    hipMemcpy(d_in, h_in, N * stride * sizeof(float), hipMemcpyHostToDevice);
    
    // Number of threads per block (multiple of 32)
    const int threadsPerBlock = 256;
    // Number of blocks per grid
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    uncoalescedCopy<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, N, stride);
    hipDeviceSynchronize();
    
    hipFree(d_in);
    hipFree(d_out);
    hipHostFree(h_in);
    hipHostFree(h_out);
    
    return 0;
}
