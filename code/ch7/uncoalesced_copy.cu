// uncoalesced_copy.cu -- demonstrate strided global memory loads.

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

__global__ void uncoalescedCopy(const float* __restrict__ in,
                                float* __restrict__ out,
                                int n,
                                int stride) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    out[idx] = in[idx * stride];
  }
}

int main() {
  constexpr int N = 1 << 20;
  constexpr int STRIDE = 2;
  static_assert(STRIDE >= 1, "Stride must be positive");
  float* h_in = nullptr;
  float* h_out = nullptr;
  CUDA_CHECK(hipHostMalloc(&h_in, N * STRIDE * sizeof(float), hipHostMallocDefault));
  CUDA_CHECK(hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault));
  for (int i = 0; i < N * STRIDE; ++i) {
    h_in[i] = static_cast<float>(i);
  }

  float *d_in = nullptr, *d_out = nullptr;
  CUDA_CHECK(hipMalloc(&d_in, N * STRIDE * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_out, N * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_in, h_in, N * STRIDE * sizeof(float), hipMemcpyHostToDevice));

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  uncoalescedCopy<<<grid, block>>>(d_in, d_out, N, STRIDE);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost));
  std::printf("out[0]=%.1f out[last]=%.1f\n", h_out[0], h_out[N - 1]);

  CUDA_CHECK(hipFree(d_in));
  CUDA_CHECK(hipFree(d_out));
  CUDA_CHECK(hipHostFree(h_in));
  CUDA_CHECK(hipHostFree(h_out));
  return 0;
}
