#include <hip/hip_runtime.h>
#include <iostream>

__global__ void coalescedCopy(const float* __restrict__ in,
                             float* __restrict__ out,
                             int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        // Contiguous load
        // Threads copy neighboring elements
        out[idx] = in[idx];
    }
}

int main() {
    const int n = 1 << 20;
    const size_t bytes = n * sizeof(float);
    
    // 1) Allocate pinned host memory
    float *h_in = nullptr, *h_out = nullptr;
    hipHostMalloc(&h_in, bytes, hipHostMallocDefault); // page-locked host alloc
    hipHostMalloc(&h_out, bytes, hipHostMallocDefault);
    
    // 2) Initialize input
    for (int i = 0; i < n; ++i) {
        h_in[i] = static_cast<float>(i);
    }
    
    // 3) Allocate device memory
    float *d_in = nullptr, *d_out = nullptr;
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, bytes);
    
    // 4) Copy to device
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
    
    // 5) Launch kernel
    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);
    coalescedCopy<<<grid, block>>>(d_in, d_out, n);
    hipDeviceSynchronize();
    
    // 6) Copy back to host
    hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);
    
    // 7) Clean up
    hipFree(d_in);
    hipFree(d_out);
    hipHostFree(h_in);
    hipHostFree(h_out);
    
    return 0;
}
