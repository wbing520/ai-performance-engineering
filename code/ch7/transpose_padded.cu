#include "hip/hip_runtime.h"
// Architecture-specific optimizations for CUDA 12.9
// Targets Blackwell B200/B300 (sm_100)
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#define TILE_DIM 32
#define PAD 1 // padding columns to avoid bank conflicts

__global__ void transposePadded(const float *idata, float *odata, int width) {
    // Each row is TILE_DIM+1 elements to shift bank mapping
    __shared__ float tile[TILE_DIM][TILE_DIM + PAD];
    cg::thread_block block = cg::this_thread_block();
    
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    
    tile[threadIdx.x][threadIdx.y] = idata[y * width + x];
    
    block.sync();
    
    odata[x * width + y] = tile[threadIdx.y][threadIdx.x];
}

int main() {
    const int N = 1024;
    size_t size = N * N * sizeof(float);
    
    float *h_idata = (float*)malloc(size);
    float *h_odata = (float*)malloc(size);
    
    // Initialize input data
    for (int i = 0; i < N * N; ++i) {
        h_idata[i] = static_cast<float>(i);
    }
    
    float *d_idata, *d_odata;
    hipMalloc(&d_idata, size);
    hipMalloc(&d_odata, size);
    
    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);
    
    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid(N / TILE_DIM, N / TILE_DIM);
    
    transposePadded<<<grid, block>>>(d_idata, d_odata, N);
    hipDeviceSynchronize();
    
    // Copy result back and verify (optional)
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);
    
    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);
    
    return 0;
}
