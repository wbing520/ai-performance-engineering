#include "hip/hip_runtime.h"
// unified_memory.cu
// Minimal example using CUDA managed memory with prefetching.

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float* data, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    data[idx] = data[idx] * data[idx] + 1.0f;
  }
}

int main() {
  constexpr int N = 1 << 20;
  size_t bytes = N * sizeof(float);

  float* data = nullptr;
  hipMallocManaged(&data, bytes);

  for (int i = 0; i < N; ++i) {
    data[i] = static_cast<float>(i);
  }

  int device = 0;
  hipGetDevice(&device);
  hipMemPrefetchAsync(data, bytes, device);

  int block = 256;
  int grid = (N + block - 1) / block;
  kernel<<<grid, block>>>(data, N);
  hipDeviceSynchronize();

  hipMemPrefetchAsync(data, bytes, hipCpuDeviceId);
  hipDeviceSynchronize();

  printf("First value: %.1f\n", data[0]);

  hipFree(data);
  return 0;
}
