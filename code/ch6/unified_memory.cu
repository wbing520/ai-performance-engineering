#include "hip/hip_runtime.h"
// unified_memory.cu
// Example demonstrating CUDA Managed Memory (Unified Memory)

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void unifiedMemoryKernel(float* data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = data[idx] * data[idx] + 1.0f;
    }
}

int main() {
    const int N = 1024 * 1024;
    float *data;
    
    // Allocate unified memory accessible from both CPU and GPU
    hipMallocManaged(&data, N * sizeof(float));
    
    // Initialize data on CPU
    for (int i = 0; i < N; ++i) {
        data[i] = float(i);
    }
    
    // Get GPU device ID for prefetching
    int device;
    hipGetDevice(&device);
    
    // Prefetch data to GPU before kernel launch
    hipMemPrefetchAsync(data, N * sizeof(float), device);
    
    // Give memory advice
    hipMemAdvise(data, N * sizeof(float), hipMemAdviseSetPreferredLocation, device);
    hipMemAdvise(data, N * sizeof(float), hipMemAdviseSetReadMostly, device);
    
    // Launch kernel
    int blocks = (N + 255) / 256;
    int threads = 256;
    unifiedMemoryKernel<<<blocks, threads>>>(data, N);
    
    // Wait for kernel completion
    hipDeviceSynchronize();
    
    // Prefetch data back to CPU for reading
    hipMemPrefetchAsync(data, N * sizeof(float), hipCpuDeviceId);
    hipDeviceSynchronize();
    
    // Access data on CPU
    printf("First 5 results: %.1f %.1f %.1f %.1f %.1f\n", 
           data[0], data[1], data[2], data[3], data[4]);
    
    // Free unified memory
    hipFree(data);
    
    return 0;
}
