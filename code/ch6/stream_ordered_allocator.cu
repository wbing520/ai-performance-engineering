// stream_ordered_allocator.cu
// Minimal example of cudaMallocAsync + streams.

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(const float* inp, float* out, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    out[idx] = inp[idx] * inp[idx] + 1.0f;
  }
}

int main() {
  constexpr int N = 1 << 20;
  size_t bytes = N * sizeof(float);

  float *h_in, *h_out;
  hipHostMalloc(&h_in, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_out, bytes, hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    h_in[i] = static_cast<float>(i);
  }

  hipStream_t stream;
  hipStreamCreate(&stream);

  float *d_in, *d_out;
  hipMallocAsync(&d_in, bytes, stream);
  hipMallocAsync(&d_out, bytes, stream);

  hipMemcpyAsync(d_in, h_in, bytes, hipMemcpyHostToDevice, stream);

  int block = 256;
  int grid = (N + block - 1) / block;
  kernel<<<grid, block, 0, stream>>>(d_in, d_out, N);

  hipMemcpyAsync(h_out, d_out, bytes, hipMemcpyDeviceToHost, stream);
  hipStreamSynchronize(stream);

  printf("Result[0]=%.1f\n", h_out[0]);

  hipFreeAsync(d_in, stream);
  hipFreeAsync(d_out, stream);
  hipStreamDestroy(stream);
  hipHostFree(h_in);
  hipHostFree(h_out);
  return 0;
}
