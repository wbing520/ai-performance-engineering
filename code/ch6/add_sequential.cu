// addSequential.cu
// Sequential vector addition example (poor performance)

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1'000'000;

// Single thread does all N additions
__global__ void addSequential(const float* A, const float* B, float* C, int N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < N; ++i) {
            C[i] = A[i] + B[i];
        }
    }
}

int main() {
    // Allocate and initialize host
    float* h_A = nullptr;
    float* h_B = nullptr;
    float* h_C = nullptr;
    hipHostMalloc(&h_A, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < N; ++i) {
        h_A[i] = float(i);
        h_B[i] = float(i * 2);
    }
    
    // Allocate device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));
    
    // Copy inputs to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Time the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    // Launch: one thread
    addSequential<<<1,1>>>(d_A, d_B, d_C, N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Ensure completion before exit
    hipDeviceSynchronize();
    
    // Copy results back to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("Sequential kernel time: %.2f ms\n", milliseconds);
    printf("Result: C[0] = %.1f, C[N-1] = %.1f\n", h_C[0], h_C[N-1]);
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
