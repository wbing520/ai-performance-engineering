// add_sequential.cu
// Naive sequential CUDA example for Chapter 6 (illustrates poor GPU utilization).

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1'000'000;

__global__ void addSequential(const float* A, const float* B, float* C, int n) {
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    for (int i = 0; i < n; ++i) {
      C[i] = A[i] + B[i];
    }
  }
}

int main() {
  float *h_A, *h_B, *h_C;
  hipHostMalloc(&h_A, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_B, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_C, N * sizeof(float), hipHostMallocDefault);

  for (int i = 0; i < N; ++i) {
    h_A[i] = static_cast<float>(i);
    h_B[i] = static_cast<float>(2 * i);
  }

  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, N * sizeof(float));
  hipMalloc(&d_B, N * sizeof(float));
  hipMalloc(&d_C, N * sizeof(float));

  hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

  addSequential<<<1, 1>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();

  hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("C[0]=%.1f, C[N-1]=%.1f\n", h_C[0], h_C[N - 1]);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipHostFree(h_A);
  hipHostFree(h_B);
  hipHostFree(h_C);
  return 0;
}
