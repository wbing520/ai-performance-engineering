// addParallel.cu
// Parallel vector addition example (optimal performance)

#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 1'000'000;

// One thread per element
__global__ void addParallel(const float* __restrict__ A,
                           const float* __restrict__ B,
                           float* __restrict__ C,
                           int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    // Allocate and initialize host
    float* h_A = nullptr;
    float* h_B = nullptr;
    float* h_C = nullptr;
    hipHostMalloc(&h_A, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, N * sizeof(float), hipHostMallocDefault);
    
    for (int i = 0; i < N; ++i) {
        h_A[i] = float(i);
        h_B[i] = float(i * 2);
    }
    
    // Allocate device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));
    
    // Copy inputs to device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);
    
    // Time the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    // Configure and launch: many threads
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    addParallel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Ensure completion before exit
    hipDeviceSynchronize();
    
    // Copy results back to host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("Parallel kernel time: %.2f ms\n", milliseconds);
    printf("Result: C[0] = %.1f, C[N-1] = %.1f\n", h_C[0], h_C[N-1]);
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
