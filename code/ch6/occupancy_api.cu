// occupancy_api.cu -- demonstrate cudaOccupancyMaxPotentialBlockSize (CUDA 12.9).

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

__global__ void sampleKernel(float* data, int n) {
  extern __shared__ float tile[];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    tile[threadIdx.x] = data[idx];
    __syncthreads();
    data[idx] = sqrtf(tile[threadIdx.x] * tile[threadIdx.x] + 1.0f);
  }
}

int main() {
  constexpr int N = 1 << 20;
  float* h_data = new float[N];
  for (int i = 0; i < N; ++i) {
    h_data[i] = static_cast<float>(i % 1000) / 1000.0f;
  }

  float* d_data = nullptr;
  CUDA_CHECK(hipMalloc(&d_data, N * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice));

  int min_grid = 0;
  int block_size = 0;
  CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(
      &min_grid,
      &block_size,
      sampleKernel,
      block_size * sizeof(float),
      0));

  std::printf("Suggested block size: %d\n", block_size);
  std::printf("Minimum grid for full occupancy: %d\n", min_grid);

  int grid = (N + block_size - 1) / block_size;
  if (grid < min_grid) grid = min_grid;

  sampleKernel<<<grid, block_size, block_size * sizeof(float)>>>(d_data, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost));
  std::printf("First result = %.3f\n", h_data[0]);

  CUDA_CHECK(hipFree(d_data));
  delete[] h_data;
  return 0;
}
