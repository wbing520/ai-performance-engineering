// atomic_work_queue.cu -- dynamic work distribution with error checks.

#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstdio>
#include <vector>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t status = (call);                                             \
    if (status != hipSuccess) {                                             \
      std::fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,     \
                    hipGetErrorString(status));                            \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

__device__ unsigned int g_index = 0;

__global__ void compute_static(const float* input, float* output, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    int work = idx & 255;
    float sum = 0.0f;
    for (int i = 0; i < work; ++i) {
      sum += sinf(input[idx]) * cosf(input[idx]);
    }
    output[idx] = sum;
  }
}

__global__ void compute_dynamic(const float* input, float* output, int n) {
  unsigned mask = __activemask();
  int lane = threadIdx.x & 31;
  while (true) {
    unsigned base = 0;
    if (lane == 0) {
      base = atomicAdd(&g_index, 32);
    }
    base = __shfl_sync(mask, base, 0);
    unsigned idx = base + lane;
    if (idx >= (unsigned)n) break;
    int work = idx & 255;
    float sum = 0.0f;
    for (int i = 0; i < work; ++i) {
      sum += sinf(input[idx]) * cosf(input[idx]);
    }
    output[idx] = sum;
  }
}

static void reset_counter() {
  unsigned zero = 0;
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_index), &zero, sizeof(unsigned)));
}

int main() {
  constexpr int N = 1 << 20;
  std::vector<float> h_in(N);
  for (int i = 0; i < N; ++i) h_in[i] = float(i) / N;
  float *d_in = nullptr, *d_out = nullptr;
  CUDA_CHECK(hipMalloc(&d_in, N * sizeof(float)));
  CUDA_CHECK(hipMalloc(&d_out, N * sizeof(float)));
  CUDA_CHECK(hipMemcpy(d_in, h_in.data(), N * sizeof(float), hipMemcpyHostToDevice));

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);

  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start));
  compute_static<<<grid, block>>>(d_in, d_out, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  float static_ms = 0.0f;
  CUDA_CHECK(hipEventElapsedTime(&static_ms, start, stop));

  reset_counter();
  CUDA_CHECK(hipEventRecord(start));
  compute_dynamic<<<grid, block>>>(d_in, d_out, N);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));
  float dynamic_ms = 0.0f;
  CUDA_CHECK(hipEventElapsedTime(&dynamic_ms, start, stop));

  std::printf("Static: %.2f ms\nDynamic: %.2f ms (speedup %.2f x)\n", static_ms, dynamic_ms, static_ms / dynamic_ms);

  CUDA_CHECK(hipFree(d_in));
  CUDA_CHECK(hipFree(d_out));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  return 0;
}
