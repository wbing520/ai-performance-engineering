#include <hip/hip_runtime.h>
#define TILE_DIM 32

__global__ void transposeNaive(const float *idata, float *odata, int width) {
    __shared__ float tile[TILE_DIM][TILE_DIM];
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    tile[threadIdx.y][threadIdx.x] = idata[y * width + x];
    __syncthreads();
    odata[x * width + y] = tile[threadIdx.x][threadIdx.y];
}

int main() {
    const int N = 1024;
    size_t size = N * N * sizeof(float);
    float *h_idata = (float*)malloc(size), *h_odata = (float*)malloc(size);
    float *d_idata, *d_odata;
    hipMalloc(&d_idata, size); hipMalloc(&d_odata, size);
    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);
    dim3 block(TILE_DIM, TILE_DIM), grid(N/TILE_DIM, N/TILE_DIM);
    transposeNaive<<<grid, block>>>(d_idata, d_odata, N);
    hipDeviceSynchronize();
    hipFree(d_idata); hipFree(d_odata);
    free(h_idata); free(h_odata);
    return 0;
}
