// add.cu — CUDA 13.0, C++17
#include <hip/hip_runtime.h>
#include <iostream>

// Elementwise addition kernel
__global__ void vecAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 1 << 20; // 1M elements
    size_t bytes = N * sizeof(float);

    // Host
    float *h_A = new float[N], *h_B = new float[N], *h_C = new float[N];
    for (int i = 0; i < N; ++i) {
        h_A[i] = float(i);
        h_B[i] = float(N - i);
    }

    // Device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    vecAdd<<<grid, block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}
