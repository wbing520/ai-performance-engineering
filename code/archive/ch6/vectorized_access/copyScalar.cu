#include <hip/hip_runtime.h>

__global__ void copyScalar(const float* __restrict__ in, float* __restrict__ out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = in[idx];
    }
}

int main() {
    const int N = 1 << 20;
    float *h_in = new float[N], *h_out = new float[N];
    for(int i = 0; i < N; ++i) h_in[i] = float(i);

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256, blocks = (N + threads - 1) / threads;
    copyScalar<<<blocks, threads>>>(d_in, d_out, N);
    hipDeviceSynchronize();

    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
