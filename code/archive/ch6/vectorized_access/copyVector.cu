#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
struct float4 { float x, y, z, w; };

__global__ void copyVector(const float4* __restrict__ in, float4* __restrict__ out, int N4) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N4) {
        out[idx] = in[idx];
    }
}

int main() {
    const int N4 = (1 << 20) / 4;
    float4 *h_in = new float4[N4], *h_out = new float4[N4];
    for(int i = 0; i < N4; ++i) {
        h_in[i] = {float(4*i), float(4*i+1), float(4*i+2), float(4*i+3)};
    }

    float4 *d_in, *d_out;
    hipMalloc(&d_in, N4 * sizeof(float4));
    hipMalloc(&d_out, N4 * sizeof(float4));
    hipMemcpy(d_in, h_in, N4 * sizeof(float4), hipMemcpyHostToDevice);

    int threads = 256, blocks = (N4 + threads - 1) / threads;
    copyVector<<<blocks, threads>>>(d_in, d_out, N4);
    hipDeviceSynchronize();

    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
