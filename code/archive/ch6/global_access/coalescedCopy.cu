#include <hip/hip_runtime.h>
#include <iostream>

__global__ void coalescedCopy(const float* __restrict__ in, float* __restrict__ out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx];
    }
}

int main() {
    const int n = 1 << 20;
    float *h_in = new float[n], *h_out = new float[n];
    for(int i = 0; i < n; ++i) h_in[i] = float(i);

    float *d_in, *d_out;
    hipMalloc(&d_in, n * sizeof(float));
    hipMalloc(&d_out, n * sizeof(float));
    hipMemcpy(d_in, h_in, n * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256, blocks = (n + threads - 1) / threads;
    coalescedCopy<<<blocks, threads>>>(d_in, d_out, n);
    hipDeviceSynchronize();

    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
