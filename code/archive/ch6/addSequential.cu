#include <hip/hip_runtime.h>
#include <iostream>
const int N = 1'000'000;

// Single thread does all N additions
__global__ void addSequential(const float* A, const float* B, float* C, int N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < N; ++i) {
            C[i] = A[i] + B[i];
        }
    }
}

int main() {
    // Allocate and initialize host data
    float *h_A = nullptr, *h_B = nullptr, *h_C = nullptr;
    hipHostMalloc(&h_A, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_B, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_C, N * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < N; ++i) {
        h_A[i] = float(i);
        h_B[i] = float(i * 2);
    }
    // Allocate device data
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);
    // Launch one thread (one block of one thread)
    addSequential<<<1, 1>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    // Copy result back
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
    return 0;
}
