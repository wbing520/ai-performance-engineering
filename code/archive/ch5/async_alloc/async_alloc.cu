#include <hip/hip_runtime.h>
#include <iostream>

__global__ void myKernel(float* buf, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) buf[idx] = buf[idx] * 2.0f;
}

int main() {
    hipStream_t s;
    hipStreamCreate(&s);

    const int N = 1<<20;
    float *d_buf;
    hipMallocAsync(&d_buf, N * sizeof(float), s);

    int threads = 256, blocks = (N + threads-1)/threads;
    myKernel<<<blocks, threads, 0, s>>>(d_buf, N);

    hipFreeAsync(d_buf, s);
    hipStreamSynchronize(s);
    hipStreamDestroy(s);
    return 0;
}
