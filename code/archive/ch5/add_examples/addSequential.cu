#include <hip/hip_runtime.h>
const int N = 1000000;

__global__ void addSequential(const float* A, const float* B, float* C, int N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for(int i = 0; i < N; ++i) C[i] = A[i] + B[i];
    }
}

int main() {
    float *h_A = new float[N], *h_B = new float[N], *h_C = new float[N];
    for(int i = 0; i < N; ++i){ h_A[i] = i; h_B[i] = 2*i; }
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);
    addSequential<<<1,1>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}
