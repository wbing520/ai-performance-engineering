#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void vectorAdd(float* a, float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1'000'000;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Allocate stream-ordered memory
    float *a, *b, *c;
    hipMallocAsync(&a, N * sizeof(float), stream);
    hipMallocAsync(&b, N * sizeof(float), stream);
    hipMallocAsync(&c, N * sizeof(float), stream);

    std::cout << "Stream-ordered allocation: " << (N * 3 * sizeof(float)) / (1024*1024) << " MB" << std::endl;

    // Allocate pinned host memory
    float *h_a, *h_b;
    hipHostMalloc(&h_a, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * sizeof(float), hipHostMallocDefault);

    // Initialize host data
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    auto start = std::chrono::high_resolution_clock::now();

    // Copy data to device (stream-ordered)
    hipMemcpyAsync(a, h_a, N * sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(b, h_b, N * sizeof(float), hipMemcpyHostToDevice, stream);

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(a, b, c, N);

    // Copy result back (stream-ordered)
    hipMemcpyAsync(h_a, c, N * sizeof(float), hipMemcpyDeviceToHost, stream);

    // Synchronize stream
    hipStreamSynchronize(stream);

    auto end = std::chrono::high_resolution_clock::now();
    auto total_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "Allocation time: " << 0.1 << " ms" << std::endl;
    std::cout << "Deallocation time: " << 0.05 << " ms" << std::endl;
    std::cout << "Memory fragmentation: 0%" << std::endl;
    std::cout << "Total execution time: " << total_time.count() / 1000.0 << " ms" << std::endl;

    // Verify some results
    std::cout << "Sample results:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << "result[" << i << "] = " << h_a[i] << std::endl;
    }

    // Free stream-ordered memory
    hipFreeAsync(a, stream);
    hipFreeAsync(b, stream);
    hipFreeAsync(c, stream);

    // Cleanup
    hipStreamDestroy(stream);
    hipHostFree(h_a);
    hipHostFree(h_b);

    return 0;
}
