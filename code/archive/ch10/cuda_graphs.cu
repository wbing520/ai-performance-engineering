#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void vectorMultiply(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    const int N = 1'000'000;
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory
    float *h_a, *h_b, *h_c;
    hipHostMalloc(&h_a, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_c, N * sizeof(float), hipHostMallocDefault);

    // Initialize data
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    float *d_a, *d_b, *d_c, *d_temp;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));
    hipMalloc(&d_temp, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Regular execution (not using graphs)
    auto start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < 100; ++i) {
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a, d_b, d_temp, N);
        vectorMultiply<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_temp, d_b, d_c, N);
    }
    
    hipStreamSynchronize(stream);
    auto end = std::chrono::high_resolution_clock::now();
    auto regular_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    // Create CUDA Graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a, d_b, d_temp, N);
    vectorMultiply<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_temp, d_b, d_c, N);
    
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);

    // Graph execution
    start = std::chrono::high_resolution_clock::now();
    
    for (int i = 0; i < 100; ++i) {
        hipGraphLaunch(graphExec, stream);
    }
    
    hipStreamSynchronize(stream);
    end = std::chrono::high_resolution_clock::now();
    auto graph_time = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "CUDA Graph created successfully" << std::endl;
    std::cout << "Graph execution time: " << graph_time.count() / 1000.0 << " ms" << std::endl;
    std::cout << "Regular execution time: " << regular_time.count() / 1000.0 << " ms" << std::endl;
    std::cout << "Speedup: " << (double)regular_time.count() / graph_time.count() << "x" << std::endl;

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); hipFree(d_temp);
    hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);

    return 0;
}
