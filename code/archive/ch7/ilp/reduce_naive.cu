#include "hip/hip_runtime.h"
// reduce_naive.cu
// Hardware: GB200/H100, CUDA 13.0, C++17, Python 3.11, OpenAI Triton 2.5.0
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void reduce_naive(const float* data, float* out, int N) {
    __shared__ float smem[256];
    int tid = threadIdx.x;
    float sum = 0.0f;
    for(int i=tid;i<N;i+=blockDim.x) sum += data[i];
    smem[tid] = sum;
    __syncthreads();
    for(int s=blockDim.x/2; s>0; s>>=1) {
        if(tid<s) smem[tid]+=smem[tid+s];
        __syncthreads();
    }
    if(tid==0) out[blockIdx.x]=smem[0];
}

int main(){
    const int N = 1<<20;
    float *h_data=new float[N];
    for(int i=0;i<N;i++) h_data[i]=1.0f;
    float *d_data,*d_out;
    hipMalloc(&d_data,N*sizeof(float));
    hipMalloc(&d_out,sizeof(float));
    hipMemcpy(d_data,h_data,N*sizeof(float),hipMemcpyHostToDevice);

    reduce_naive<<<1,256>>>(d_data,d_out,N);
    hipDeviceSynchronize();

    float result;
    hipMemcpy(&result,d_out,sizeof(float),hipMemcpyDeviceToHost);
    std::cout<<"Sum = "<<result<<"
";
    hipFree(d_data);hipFree(d_out);
    delete[] h_data;
    return 0;
}