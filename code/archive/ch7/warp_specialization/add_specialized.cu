#include "hip/hip_runtime.h"
// add_specialized.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_specialized(const float* A, const float* B, float* C, int N) {
    extern __shared__ float smem[];
    float* Abuf=smem;
    float* Bbuf=smem+blockDim.x;
    int warpId=threadIdx.x/32, lane=threadIdx.x%32;
    for(int base=blockIdx.x*blockDim.x; base<N; base+=blockDim.x){
        if(warpId==0){
            int idx=base+lane;
            if(idx<N){Abuf[lane]=A[idx];Bbuf[lane]=B[idx];}
        }
        __syncthreads();
        if(warpId>0){
            int idx=base+(warpId-1)*32+lane;
            if(idx<N) C[idx]=Abuf[lane]+Bbuf[lane];
        }
        __syncthreads();
    }
}

int main(){
    const int N=1<<20;
    float *h_A=new float[N],*h_B=new float[N],*h_C=new float[N];
    for(int i=0;i<N;i++){h_A[i]=1;h_B[i]=2;}
    float *d_A,*d_B,*d_C;
    hipMalloc(&d_A,N*sizeof(float));
    hipMalloc(&d_B,N*sizeof(float));
    hipMalloc(&d_C,N*sizeof(float));
    hipMemcpy(d_A,h_A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,N*sizeof(float),hipMemcpyHostToDevice);

    int threads=128;
    add_specialized<<<N/threads,threads,2*threads*sizeof(float)>>>(d_A,d_B,d_C,N);
    hipDeviceSynchronize();
    hipMemcpy(h_C,d_C,N*sizeof(float),hipMemcpyDeviceToHost);
    std::cout<<"C[0]="<<h_C[0]<<"
";
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    delete[] h_A;delete[] h_B;delete[] h_C;
    return 0;
}