#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matmul_fp32(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 1024;
    size_t size = N * N * sizeof(float);
    float *h_A = new float[N*N], *h_B = new float[N*N], *h_C = new float[N*N];
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); hipMalloc(&d_B, size); hipMalloc(&d_C, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    dim3 block(16,16), grid((N+15)/16,(N+15)/16);
    matmul_fp32<<<grid, block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}
