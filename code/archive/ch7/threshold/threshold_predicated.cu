#include <hip/hip_runtime.h>
#include <iostream>

__global__ void threshold_predicated(const float* X, float* Y, float threshold, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float x = X[i];
        float val = (x > threshold) ? x : 0.0f;
        Y[i] = val;
    }
}

int main() {
    const int N = 1 << 20;
    float *h_X = new float[N], *h_Y = new float[N];
    for (int i = 0; i < N; ++i) h_X[i] = float(rand())/RAND_MAX - 0.5f;
    float *d_X, *d_Y;
    hipMalloc(&d_X, N*sizeof(float));
    hipMalloc(&d_Y, N*sizeof(float));
    hipMemcpy(d_X, h_X, N*sizeof(float), hipMemcpyHostToDevice);
    int threads = 256, blocks = (N+threads-1)/threads;
    threshold_predicated<<<blocks, threads>>>(d_X, d_Y, 0.0f, N);
    hipDeviceSynchronize();
    hipFree(d_X); hipFree(d_Y);
    delete[] h_X; delete[] h_Y;
    return 0;
}
