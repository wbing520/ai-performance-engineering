#include <mma.h>
#include <hip/hip_runtime.h>
using namespace nvcuda;

#define TILE_DIM 16

__global__ void matmul_tensorcore(const half* A, const half* B, float* C, int N) {
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y) / 32;
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x) / 32;

    wmma::fragment<wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_DIM, TILE_DIM, TILE_DIM, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    for (int k = 0; k < N; k += TILE_DIM) {
        int aRow = warpM * TILE_DIM, aCol = k;
        int bRow = k, bCol = warpN * TILE_DIM;
        wmma::load_matrix_sync(a_frag, A + aRow * N + aCol, N);
        wmma::load_matrix_sync(b_frag, B + bRow * N + bCol, N);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    int cRow = warpM * TILE_DIM, cCol = warpN * TILE_DIM;
    wmma::store_matrix_sync(C + cRow * N + cCol, acc_frag, N, wmma::mem_row_major);
}

int main() {
    const int N = 1024;
    size_t size = N * N * sizeof(half);
    half *h_A = new half[N*N], *h_B = new half[N*N];
    float *h_C = new float[N*N];
    half *d_A, *d_B; float *d_C;
    hipMalloc(&d_A, size); hipMalloc(&d_B, size); hipMalloc(&d_C, N*N*sizeof(float));
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    dim3 block(32,32), grid((N+31)/32,(N+31)/32);
    matmul_tensorcore<<<grid, block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}
