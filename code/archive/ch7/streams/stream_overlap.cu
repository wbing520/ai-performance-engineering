// stream_overlap.cu
#include <hip/hip_runtime.h>
#include <iostream>
#define N (1<<20)

__global__ void dummy_compute(float *data) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx<N) data[idx]*=2.0f;
}

int main(){
    float *h_buf[2], *d_buf[2];
    for(int i=0;i<2;i++){
        hipHostMalloc(&h_buf[i], N*sizeof(float), hipHostMallocDefault);
        hipMalloc(&d_buf[i], N*sizeof(float));
        for(int j=0;j<N;j++) h_buf[i][j]=1.0f;
    }
    hipStream_t st_copy, st_comp;
    hipStreamCreate(&st_copy);
    hipStreamCreate(&st_comp);

    for(int i=0;i<4;i++){
        int bi=i%2, pi=(i+1)%2;
        hipMemcpyAsync(d_buf[bi],h_buf[bi],N*sizeof(float),hipMemcpyHostToDevice,st_copy);
        if(i>0){
            dummy_compute<<<(N+255)/256,256,0,st_comp>>>(d_buf[pi]);
            hipMemcpyAsync(h_buf[pi],d_buf[pi],N*sizeof(float),hipMemcpyDeviceToHost,st_copy);
        }
    }
    dummy_compute<<<(N+255)/256,256,0,st_comp>>>(d_buf[0]);
    hipMemcpyAsync(h_buf[0],d_buf[0],N*sizeof(float),hipMemcpyDeviceToHost,st_copy);

    hipStreamSynchronize(st_copy);
    hipStreamSynchronize(st_comp);
    std::cout<<"Done"<<std::endl;
    return 0;
}