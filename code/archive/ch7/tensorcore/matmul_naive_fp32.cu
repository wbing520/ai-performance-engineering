// matmul_naive_fp32.cu
// Hardware: Grace-Blackwell GB200 (Compute Capability 10.0) or fallback H100
// Software: CUDA 13.0, C++17, Nsight Systems 2025.2.1, Nsight Compute 2024.3
// Python: 3.11, PyTorch nightly 2.8.0+, OpenAI Triton 2.5.0

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matmul_naive(const float* A, const float* B, float* C, int N, int K, int M) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < M) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row*K + k] * B[k*M + col];
        }
        C[row*M + col] = sum;
    }
}

int main() {
    int N = 1024, K = 1024, M = 1024;
    size_t bytesA = N * K * sizeof(float);
    size_t bytesB = K * M * sizeof(float);
    size_t bytesC = N * M * sizeof(float);

    float *h_A = new float[N*K];
    float *h_B = new float[K*M];
    float *h_C = new float[N*M];
    for (int i = 0; i < N*K; ++i) h_A[i] = 1.0f;
    for (int i = 0; i < K*M; ++i) h_B[i] = 1.0f;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytesA);
    hipMalloc(&d_B, bytesB);
    hipMalloc(&d_C, bytesC);

    hipMemcpy(d_A, h_A, bytesA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytesB, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    matmul_naive<<<grid, block>>>(d_A, d_B, d_C, N, K, M);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytesC, hipMemcpyDeviceToHost);

    std::cout << "C[0] = " << h_C[0] << std::endl;

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}