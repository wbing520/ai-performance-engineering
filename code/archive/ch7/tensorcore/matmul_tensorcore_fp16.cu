// matmul_tensorcore_fp16.cu
// Hardware: Grace-Blackwell GB200 or H100 fallback
// Software: CUDA 13.0, C++17, Nsight Systems 2025.2.1, Nsight Compute 2024.3
// Python: 3.11, PyTorch nightly 2.8.0+, OpenAI Triton 2.5.0

#include <mma.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace nvcuda;

__global__ void matmul_tensorcore(const half* A, const half* B, float* C, int N, int K, int M) {
    int warpM = (blockIdx.y * blockDim.y + threadIdx.y) / 16;
    int warpN = (blockIdx.x * blockDim.x + threadIdx.x) / 16;

    wmma::fragment<wmma::matrix_a,16,16,16,half,wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b,16,16,16,half,wmma::col_major> bFrag;
    wmma::fragment<wmma::accumulator,16,16,16,float> cFrag;
    wmma::fill_fragment(cFrag, 0.0f);

    if(warpM * 16 < N && warpN * 16 < M) {
        wmma::load_matrix_sync(aFrag, A + warpM*16*K, K);
        wmma::load_matrix_sync(bFrag, B + warpN*16, K);
        wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        wmma::store_matrix_sync(C + warpM*16*M + warpN*16, cFrag, M, wmma::mem_row_major);
    }
}

int main() {
    int N=1024, K=1024, M=1024;
    size_t bytesA=N*K*sizeof(half);
    size_t bytesB=K*M*sizeof(half);
    size_t bytesC=N*M*sizeof(float);

    half *h_A=new half[N*K];
    half *h_B=new half[K*M];
    float *h_C=new float[N*M];
    for(int i=0;i<N*K;i++) h_A[i]=__float2half(1.0f);
    for(int i=0;i<K*M;i++) h_B[i]=__float2half(1.0f);

    half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, bytesA);
    hipMalloc(&d_B, bytesB);
    hipMalloc(&d_C, bytesC);

    hipMemcpy(d_A, h_A, bytesA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytesB, hipMemcpyHostToDevice);

    dim3 block(16,16);
    dim3 grid((M+15)/16,(N+15)/16);
    matmul_tensorcore<<<grid,block>>>(d_A,d_B,d_C,N,K,M);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, bytesC, hipMemcpyDeviceToHost);
    std::cout<<"C[0] = "<<h_C[0]<<std::endl;

    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    delete[] h_A; delete[] h_B; delete[] h_C;
    return 0;
}