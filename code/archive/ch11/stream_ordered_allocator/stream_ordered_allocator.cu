#include "hip/hip_runtime.h"
// Architecture-specific optimizations for CUDA 12.9
// Supports Hopper H100/H200 (sm_90) and Blackwell B200/B300 (sm_100)
#include <hip/hip_runtime.h>
#include <cstdio>

// Example compute kernel
__global__ void computeKernel(float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = input[idx] * 2.0f; // Simple computation
    }
}

int main() {
    // Initialize the async memory allocator
    hipMemPool_t pool;
    hipDeviceGetDefaultMemPool(&pool, 0);
    
    // Desired number of bytes to keep in pool before
    // releasing back to the OS (tune as needed)
    size_t threshold = 1024 * 1024 * 1024; // 1GB
    hipMemPoolSetAttribute(pool, hipMemPoolAttrReleaseThreshold, &threshold);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Allocate memory using stream-ordered async allocation
    void *d_data1, *d_result1;
    void *d_data2, *d_result2;
    size_t dataSizeBytes = 1024 * sizeof(float);

    // Use hipMallocAsync on a given stream (best practice in modern multi-stream apps)
    hipMallocAsync(&d_data1, dataSizeBytes, stream1);
    hipMallocAsync(&d_result1, dataSizeBytes, stream1);
    hipMallocAsync(&d_data2, dataSizeBytes, stream2);
    hipMallocAsync(&d_result2, dataSizeBytes, stream2);

    // Allocate pinned host memory for async transfers
    float *h_data1, *h_data2, *h_result1, *h_result2;
    hipHostAlloc(&h_data1, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&h_data2, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&h_result1, dataSizeBytes, hipHostMallocDefault);
    hipHostAlloc(&h_result2, dataSizeBytes, hipHostMallocDefault);

    // Initialize host data
    for (int i = 0; i < 1024; i++) {
        h_data1[i] = (float)i;
        h_data2[i] = (float)(i * 2);
    }

    // Define grid and block dimensions
    dim3 gridDim(4);
    dim3 blockDim(256);

    // Asynchronously copy first chunk and launch its kernel in stream1
    hipMemcpyAsync(d_data1, h_data1, dataSizeBytes,
                    hipMemcpyHostToDevice, stream1);
    computeKernel<<<gridDim, blockDim, 0, stream1>>>((float*)d_data1, (float*)d_result1);
    hipMemcpyAsync(h_result1, d_result1, dataSizeBytes,
                    hipMemcpyDeviceToHost, stream1);

    // In parallel, do the same on stream2
    hipMemcpyAsync(d_data2, h_data2, dataSizeBytes, hipMemcpyHostToDevice, stream2);
    computeKernel<<<gridDim, blockDim, 0, stream2>>>((float*)d_data2, (float*)d_result2);
    hipMemcpyAsync(h_result2, d_result2, dataSizeBytes,
                    hipMemcpyDeviceToHost, stream2);

    // Wait for both streams to finish
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Print some results
    printf("Stream 1 result[0]: %f\n", h_result1[0]);
    printf("Stream 2 result[0]: %f\n", h_result2[0]);

    // Cleanup
    hipFreeAsync(d_data1, stream1);
    hipFreeAsync(d_result1, stream1);
    hipFreeAsync(d_data2, stream2);
    hipFreeAsync(d_result2, stream2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    // Free host memory
    hipHostFree(h_data1);
    hipHostFree(h_data2);
    hipHostFree(h_result1);
    hipHostFree(h_result2);

    return 0;
}
