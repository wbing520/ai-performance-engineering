#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    const int N = 1<<20;
    float *d; hipMalloc(&d, N*sizeof(float));
    dim3 b(256), g((N+255)/256);

    hipGraph_t graph;
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipGraphCreate(&graph, 0);

    for(int i=0; i<10; ++i) {
        hipGraphNode_t node;
        hipKernelNodeParams params = {0};
        params.func = (void*)addKernel;
        params.gridDim = g; params.blockDim = b;
        params.kernelParams = new void*[2]{&d, &N};
        hipGraphAddKernelNode(&node, graph, nullptr, 0, &params);
    }

    hipGraphExec_t instance;
    hipGraphInstantiate(&instance, graph, nullptr, nullptr, 0);
    hipGraphLaunch(instance, stream);
    hipStreamSynchronize(stream);

    hipFree(d);
    return 0;
}

// Kernel definition reused
__global__ void addKernel(float* data, int N) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < N) data[idx] += 1.0f;
}
