#include <hip/hip_runtime.h>
#include <cmath>

// Static uneven workload per thread
__global__ void computeKernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int work = idx % 256;
        float result = 0.0f;
        for (int i = 0; i < work; ++i) {
            result += sinf(input[idx]) * cosf(input[idx]);
        }
        output[idx] = result;
    }
}

int main() {
    const int N = 1 << 20;
    size_t bytes = N * sizeof(float);
    float *h_in = new float[N], *h_out = new float[N];
    for (int i = 0; i < N; ++i) h_in[i] = float(i)/N;
    float *d_in, *d_out;
    hipMalloc(&d_in, bytes); hipMalloc(&d_out, bytes);
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
    dim3 block(256), grid((N+255)/256);
    computeKernel<<<grid, block>>>(d_in, d_out, N);
    hipDeviceSynchronize();
    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
