#include <hip/hip_runtime.h>
#include <iostream>

// Dynamic workload via atomic queue
__global__ void computeKernelDynamic(const float* input, float* output, int N) {
    __shared__ unsigned int idx;
    if (threadIdx.x == 0) idx = 0;
    __syncthreads();
    while (true) {
        unsigned int i = atomicAdd(&idx, 1);
        if (i >= N) break;
        float result = 0.0f;
        int work = i % 256;
        for (int j = 0; j < work; ++j) {
            result += sinf(input[i]) * cosf(input[i]);
        }
        output[i] = result;
    }
}

int main() {
    const int N = 1 << 20;
    size_t bytes = N * sizeof(float);
    float *h_in = new float[N], *h_out = new float[N];
    for (int i = 0; i < N; ++i) h_in[i] = float(i)/N;
    float *d_in, *d_out;
    hipMalloc(&d_in, bytes); hipMalloc(&d_out, bytes);
    hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);
    dim3 block(256), grid((N+255)/256);
    computeKernelDynamic<<<grid, block>>>(d_in, d_out, N);
    hipDeviceSynchronize();
    hipFree(d_in); hipFree(d_out);
    delete[] h_in; delete[] h_out;
    return 0;
}
