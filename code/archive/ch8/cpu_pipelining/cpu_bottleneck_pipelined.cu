#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <thread>

void cpuWork(int i) {
    volatile float x = 0;
    for(int j=0;j<10000;++j) x += sinf(j)*cosf(j);
}

__global__ void gpuWork(float* data, int N) {
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx < N) data[idx] += 1.0f;
}

int main() {
    const int N = 1<<20;
    float *d; hipMalloc(&d, N*sizeof(float));
    for(int i=0;i<100; ++i) {
        std::thread cpu_thread(cpuWork, i);
        gpuWork<<<N/256,256>>>(d,N);
        cpu_thread.join();
        hipDeviceSynchronize();
    }
    hipFree(d);
    return 0;
}
