#include <hip/hip_runtime.h>
#include <iostream>

__global__ void computeKernel(float* data, float* out, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) out[idx] = data[idx] * 2.0f;
}

int main() {
    const int N = 1 << 20;
    float *hA = new float[N], *hB = new float[N], *hC = new float[N], *hD = new float[N];
    for (int i = 0; i < N; ++i) hA[i] = i;
    float *dA, *dB, *dC, *dD;
    hipStream_t s1, s2;
    hipStreamCreate(&s1); hipStreamCreate(&s2);
    hipMallocAsync(&dA, N * sizeof(float), s1);
    hipMallocAsync(&dC, N * sizeof(float), s1);
    hipMallocAsync(&dB, N * sizeof(float), s2);
    hipMallocAsync(&dD, N * sizeof(float), s2);
    hipMemcpyAsync(dA, hA, N * sizeof(float), hipMemcpyHostToDevice, s1);
    computeKernel<<<(N + 255) / 256, 256, 0, s1>>>(dA, dC, N);
    hipMemcpyAsync(hC, dC, N * sizeof(float), hipMemcpyDeviceToHost, s1);
    hipMemcpyAsync(dB, hB, N * sizeof(float), hipMemcpyHostToDevice, s2);
    computeKernel<<<(N + 255) / 256, 256, 0, s2>>>(dB, dD, N);
    hipMemcpyAsync(hD, dD, N * sizeof(float), hipMemcpyDeviceToHost, s2);
    hipStreamSynchronize(s1); hipStreamSynchronize(s2);
    hipFreeAsync(dA, s1); hipFreeAsync(dC, s1);
    hipFreeAsync(dB, s2); hipFreeAsync(dD, s2);
    hipStreamDestroy(s1); hipStreamDestroy(s2);
    delete[] hA; delete[] hB; delete[] hC; delete[] hD;
    return 0;
}
