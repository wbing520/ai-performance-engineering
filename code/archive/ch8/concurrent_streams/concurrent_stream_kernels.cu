#include <hip/hip_runtime.h>
#include <iostream>

// Launch small kernels concurrently on two streams
__global__ void smallKernel(int* data) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    data[idx] += 1;
}

int main() {
    const int N = 1<<20;
    int *d1, *d2;
    hipMalloc(&d1, N*sizeof(int));
    hipMalloc(&d2, N*sizeof(int));
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);
    for (int i = 0; i < 100; ++i) {
        smallKernel<<<N/256,256,0,s1>>>(d1);
        smallKernel<<<N/256,256,0,s2>>>(d2);
    }
    hipDeviceSynchronize();
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
    hipFree(d1); hipFree(d2);
    return 0;
}
