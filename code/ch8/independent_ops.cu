// independent_ops.cu -- ILP demo with two independent operations per thread.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void independent_ops(const float* a, const float* b, float* out, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float x = a[idx];
    float y = b[idx];
    float u = x * x;
    float v = y * y;
    out[idx] = u + v;
  }
}

int main() {
  float *h_a, *h_b, *h_out;
  hipHostMalloc(&h_a, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_b, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_out, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    h_a[i] = static_cast<float>(i);
    h_b[i] = static_cast<float>(N - i);
  }

  float *d_a, *d_b, *d_out;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_out, N * sizeof(float));
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  independent_ops<<<grid, block>>>(d_a, d_b, d_out, N);
  hipDeviceSynchronize();

  hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("out[0]=%.1f\n", h_out[0]);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_out);
  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_out);
  return 0;
}
