// threshold_predicated.cu -- predicated version of thresholding.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void threshold_predicated(const float* x, float* y, float threshold, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    float val = x[idx];
    y[idx] = (val > threshold) ? val : 0.0f;
  }
}

int main() {
  float *h_x, *h_y;
  hipHostMalloc(&h_x, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_y, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    h_x[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  float *d_x, *d_y;
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  threshold_predicated<<<grid, block>>>(d_x, d_y, 0.5f, N);
  hipDeviceSynchronize();

  hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("y[0]=%.3f\n", h_y[0]);

  hipFree(d_x);
  hipFree(d_y);
  hipHostFree(h_x);
  hipHostFree(h_y);
  return 0;
}
