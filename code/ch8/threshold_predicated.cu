// threshold_predicated.cu -- predicated version of thresholding.

#include <hip/hip_runtime.h>
#include <cstdio>

constexpr int N = 1 << 20;

__global__ void threshold_predicated(const float* __restrict__ X,
                                     float* __restrict__ Y,
                                     float threshold,
                                     int N) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for (int i = idx; i < N; i += stride) {
    const float x = X[i];
    // Branch-free select keeps the kernel truly predicated on the data-dependent threshold check.
    Y[i] = (x > threshold) ? x : 0.0f;
  }
}

int main() {
  float *h_x, *h_y;
  hipHostMalloc(&h_x, N * sizeof(float), hipHostMallocDefault);
  hipHostMalloc(&h_y, N * sizeof(float), hipHostMallocDefault);
  for (int i = 0; i < N; ++i) {
    h_x[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  float *d_x, *d_y;
  hipMalloc(&d_x, N * sizeof(float));
  hipMalloc(&d_y, N * sizeof(float));
  hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 block(256);
  dim3 grid((N + block.x - 1) / block.x);
  threshold_predicated<<<grid, block>>>(d_x, d_y, 0.5f, N);
  hipDeviceSynchronize();

  hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
  printf("y[0]=%.3f\n", h_y[0]);

  hipFree(d_x);
  hipFree(d_y);
  hipHostFree(h_x);
  hipHostFree(h_y);
  return 0;
}
