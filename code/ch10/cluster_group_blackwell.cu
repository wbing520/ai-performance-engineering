#include "hip/hip_runtime.h"
// Blackwell-only example: CTA cluster cooperative groups
// Demonstrates cluster-wide synchronization and shared data exchange
// Requires SM100 and a GPU/driver supporting cooperative cluster launch

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

namespace cg = cooperative_groups;

// Kernel that sums values across blocks in a cluster using block-scoped shared arrays
__global__ void cluster_sum_kernel(const float *in, float *out, int elems_per_block) {
    cg::cluster_group cluster = cg::this_cluster();
    cg::thread_block cta = cg::this_thread_block();

    extern __shared__ float sdata[]; // per-CTA shared buffer

    float sum = 0.0f;
    int base = blockIdx.x * elems_per_block;
    for (int i = threadIdx.x; i < elems_per_block; i += blockDim.x) {
        sum += in[base + i];
    }

    sdata[threadIdx.x] = sum;
    cta.sync();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            sdata[threadIdx.x] += sdata[threadIdx.x + stride];
        }
        cta.sync();
    }

    if (threadIdx.x == 0) {
        out[blockIdx.x] = sdata[0];
    }

    cluster.sync();

    if (cluster.block_rank() == 0 && threadIdx.x == 0) {
        int cluster_blocks = cluster.dim_blocks().x * cluster.dim_blocks().y * cluster.dim_blocks().z;
        int cluster_start = blockIdx.x - cluster.block_rank();

        float cluster_total = 0.0f;
        for (int b = 0; b < cluster_blocks && (cluster_start + b) < gridDim.x; ++b) {
            cluster_total += out[cluster_start + b];
        }
        out[cluster_start] = cluster_total;
    }
}

int main() {
    constexpr int cluster_size = 2;
    int num_blocks = 8; // total CTAs in the grid (must be >= cluster_size)
    int elems_per_block = 1 << 20; // 1M elements per block
    int threads = 256;
    size_t total_elems = size_t(num_blocks) * elems_per_block;
    size_t bytes = total_elems * sizeof(float);

    float *d_in = nullptr, *d_out = nullptr;
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, num_blocks * sizeof(float));

    std::vector<float> h_in(total_elems, 1.0f);
    hipMemcpy(d_in, h_in.data(), bytes, hipMemcpyHostToDevice);
    hipMemset(d_out, 0, num_blocks * sizeof(float));

    hipLaunchConfig_t cfg{};
    cfg.gridDim = dim3(num_blocks, 1, 1);
    cfg.blockDim = dim3(threads, 1, 1);
    cfg.dynamicSmemBytes = threads * sizeof(float);

    hipLaunchAttribute attr[1];
    attr[0].id = cudaLaunchAttributeClusterDimension;
    attr[0].val.clusterDim.x = cluster_size;
    attr[0].val.clusterDim.y = 1;
    attr[0].val.clusterDim.z = 1;

    cfg.attrs = attr;
    cfg.numAttrs = 1;

    hipFuncSetAttribute(reinterpret_cast<const void*>(cluster_sum_kernel), cudaFuncAttributeNonPortableClusterSizeAllowed, 1);

    hipError_t err = cudaLaunchKernelEx(&cfg, cluster_sum_kernel, d_in, d_out, elems_per_block);
    if (err != hipSuccess) {
        printf("Cluster launch not supported or failed: %s\n", hipGetErrorString(err));
        hipFree(d_in);
        hipFree(d_out);
        return 0;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel error: %s\n", hipGetErrorString(err));
        hipFree(d_in);
        hipFree(d_out);
        return -1;
    }

    std::vector<float> h_out(num_blocks, 0.0f);
    hipMemcpy(h_out.data(), d_out, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    double expected_block = static_cast<double>(elems_per_block);
    double expected_cluster_total = expected_block * cluster_size;

    printf("cluster_group_blackwell completed.\n");
    int sample_block = (num_blocks > 1) ? 1 : 0;
    printf(" - Partial sum (block %d): %.2f (expected %.2f)\n", sample_block, h_out[sample_block], expected_block);
    printf(" - Cluster total (cluster 0): %.2f (expected %.2f)\n", h_out[0], expected_cluster_total);

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
