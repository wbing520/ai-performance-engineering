#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>

static bool checkPeerAccess(int devA, int devB) {
	int canAccess = 0;
	hipDeviceCanAccessPeer(&canAccess, devA, devB);
	return canAccess != 0;
}

static float measureP2P(int devSrc, int devDst, size_t bytes, int iters) {
	hipSetDevice(devSrc);
	void *src = nullptr;
	hipMalloc(&src, bytes);
	hipMemset(src, 0, bytes);

	hipSetDevice(devDst);
	void *dst = nullptr;
	hipMalloc(&dst, bytes);
	hipMemset(dst, 0, bytes);

	hipStream_t stream;
	hipStreamCreate(&stream);

	// Warmup
	for (int i = 0; i < 3; ++i) {
		hipMemcpyPeerAsync(dst, devDst, src, devSrc, bytes, stream);
	}
	hipStreamSynchronize(stream);

	// Timed iterations
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, stream);
	for (int i = 0; i < iters; ++i) {
		hipMemcpyPeerAsync(dst, devDst, src, devSrc, bytes, stream);
	}
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);

	float ms = 0.0f;
	hipEventElapsedTime(&ms, start, stop);
	float avgMs = ms / iters;
	float gbps = (bytes / 1e9f) / (avgMs / 1e3f);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream);

	hipFree(dst);
	hipSetDevice(devSrc);
	hipFree(src);

	return gbps;
}

int main() {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount < 2) {
		printf("Need at least 2 GPUs for NVLink C2C demo. Found %d\n", deviceCount);
		return 0;
	}

	// Enable peer access where possible
	for (int i = 0; i < deviceCount; ++i) {
		for (int j = 0; j < deviceCount; ++j) {
			if (i == j) continue;
			if (checkPeerAccess(i, j)) {
				hipSetDevice(i);
				hipDeviceEnablePeerAccess(j, 0);
			}
		}
	}

	// Test large transfers to highlight Blackwell NVLink-C2C capability
	size_t bytes = size_t(1) << 30; // 1 GiB
	int iters = 10;

	for (int src = 0; src < deviceCount; ++src) {
		hipDeviceProp_t propS{};
		hipGetDeviceProperties(&propS, src);
		for (int dst = 0; dst < deviceCount; ++dst) {
			if (src == dst) continue;
			hipDeviceProp_t propD{};
			hipGetDeviceProperties(&propD, dst);
			if (!checkPeerAccess(src, dst)) {
				printf("%s -> %s: Peer access not available\n", propS.name, propD.name);
				continue;
			}
			float gbps = measureP2P(src, dst, bytes, iters);
			printf("%s -> %s: %.2f GB/s (1 GiB avg over %d iters)\n", propS.name, propD.name, gbps, iters);
		}
	}

	return 0;
}


